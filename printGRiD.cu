/***
nvcc -std=c++11 -o printGRiD.exe printGRiD.cu -gencode arch=compute_86,code=sm_86
***/

#include <random>
#include <algorithm>
#include "grid.cuh"
#define RANDOM_MEAN 0
#define RANDOM_STDEV 1
std::default_random_engine randEng(1337); // fixed seed
std::normal_distribution<double> randDist(RANDOM_MEAN, RANDOM_STDEV); //mean followed by stdiv
template <typename T>
T getRand(){return static_cast<T>(randDist(randEng));}

template <typename T>
__host__
void test(){
	T gravity = static_cast<T>(9.81);
	dim3 dimms(grid::SUGGESTED_THREADS,1,1);
	hipStream_t *streams = grid::init_grid<T>();
	grid::robotModel<T> *d_robotModel = grid::init_robotModel<T>();
	grid::gridData<T> *hd_data = grid::init_gridData<T,1>();
	
	// load q,qd,u
	for(int j = 0; j < grid::NUM_JOINTS; j++){
		hd_data->h_q_qd_u[j] = getRand<double>(); 
		hd_data->h_q_qd_u[j+grid::NUM_JOINTS] = getRand<double>(); 
		hd_data->h_q_qd_u[j+2*grid::NUM_JOINTS] = getRand<double>();
	}
	gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	printf("q,qd,u\n");
	printMat<T,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
	printMat<T,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
	printMat<T,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

	printf("eePos\n");
	grid::end_effector_positions<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
	printMat<T,1,6*grid::NUM_EES>(hd_data->h_eePos,1);

	printf("deePos\n");
	grid::end_effector_positions_gradient<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
	printMat<T,6,grid::NUM_EES*grid::NUM_JOINTS>(hd_data->h_deePos,6);

	printf("c\n");
	grid::inverse_dynamics<T,false,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printMat<T,1,grid::NUM_JOINTS>(hd_data->h_c,1);

	printf("Minv\n");
	grid::direct_minv<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
	printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_Minv,grid::NUM_JOINTS);

	printf("qdd\n");
	grid::forward_dynamics<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printMat<T,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);

	grid::inverse_dynamics_gradient<T,true,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printf("dc_dq\n");
	printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_dc_du,grid::NUM_JOINTS);
	printf("dc_dqd\n");
	printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_dc_du[grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);

	grid::forward_dynamics_gradient<T,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printf("df_dq\n");
	printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_df_du,grid::NUM_JOINTS);
	printf("df_dqd\n");
	printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_df_du[grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);

	grid::close_grid<T>(streams,d_robotModel,hd_data);
}

int main(void){
	test<float>(); return 0;
}